#include <stdlib.h>
#include <stdio.h>
#include "err.cuh"

void checkCudaSyncErr() {
    hipError_t syncErr = hipGetLastError();
    hipError_t asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printCudaSyncErr(syncErr);
    if (asyncErr != hipSuccess) printCudaAsyncErr(asyncErr);
}

void printErr(int flag) {
    switch (flag) {
        case 1:
            fprintf(stderr, "Error: could not allocate memory\n");
            exit(1);
        case 2:
            fprintf(stderr, "Error: No filename found\n");
            exit(2);
    }
}

void printCudaErr(hipError_t err) {
    fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
    exit(1);
}

void printCudaAsyncErr(hipError_t err) {
    fprintf(stderr, "Error (Async): %s\n", hipGetErrorString(err));
    exit(1);
}

void printCudaSyncErr(hipError_t err) {
    fprintf(stderr, "Error (Sync): %s\n", hipGetErrorString(err));
    exit(1);
}
