#include <stdlib.h>
#include <stdio.h>
#include "err.h"

void checkCudaSyncErr() {
    hipError_t syncErr = hipGetLastError();
    hipError_t asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) CUDAERROR(syncErr);
    if (asyncErr != hipSuccess) CUDAERROR(asyncErr);
}

void printErr(int flag) {
    switch (flag) {
        case 1:
            fprintf(stderr, "Error: could not allocate memory\n");
            exit(1);
        case 2:
            fprintf(stderr, "Error: No filename found\n");
            exit(2);
    }
}

void printCudaErr(hipError_t err) {
    fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
    exit(1);
}
