#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "erfinv.h"
#include "err.h"

__managed__ float *uniform;
__managed__ float **invGaus;
float alpha;
int numElem;
float J;
float W;

// Gaussian inverse CDF
float gaussianICDF(float p, float d) {
    return my_erfinvf(2.0*p-1.0)*J*sqrtf(2.0)/powf(d,alpha);
}

// Generate standard uniform distribution
void gererateSUD(float a, float j, float w, int num) {
    hipError_t err = hipMallocManaged(&uniform, sizeof(float)*numElem);
    if (err != hipSuccess) CUDAERROR(err);
    alpha = a;
    J = j;
    W = w;
    numElem = num;

    for (int i = 0; i < numElem; i++) {
        uniform[i] = (float) i/(float) (numElem-1);
    }
}

// Generate ICDF values
void generateICDF() {
    int r;
    hipError_t err = hipMallocManaged(&invGaus, sizeof(float*)*numElem);
    if (err != hipSuccess) CUDAERROR(err);
    for (int i = 0; i < numElem; i++) {
        err = hipMallocManaged(&invGaus[i], sizeof(float)*numElem);
        if (err != hipSuccess) CUDAERROR(err);
        for (int j = 0; j < numElem; j++) {
            r = rand()%numElem;
            invGaus[i][j] = gaussianICDF(uniform[r], i);
        }
    }
}

void freeDistributions() {
    hipFree(uniform);
    for (int i = 0; i < numElem; i++) {
        hipFree(invGaus[i]);
    }
    hipFree(invGaus);
}

__global__ void generateMaster(hiprandState_t* states, float** master) {
    int i;
    int j;
    int r;
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    if (id < numElem) {
        i = threadIndex[id]->x;
        j = threadIndex[id]->y;
        if (j >= N-i || id >= numElem || i >= N) {
            printf("id = %d (%d,%d) (%d)\n",id, i, j, N-i);
        }
        if (j == 0) {
            master[i][0] = (float)hiprand(&states[id])/((float)RAND_MAX/(float)W);
        } else {
            r = hiprand(&states[id])%numElem;
            master[i][j] = invGaus[abs(i-j)][r];
        }
    }
}
