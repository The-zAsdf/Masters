#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include "erfinv.h"

__managed__ float *uniform;
__managed__ float **invGaus;
int s;
float j0;
float alpha;

// Gaussian inverse CDF
float gaussianICDF(float p, float d) {
    return my_erfinvf(2.0*p-1.0)*j0*sqrtf(2.0)/powf(d,alpha);
}

// Generate standard uniform distribution
void gererateSUD(int size, float j, float a) {
    hipError_t err = hipMallocManaged(&uniform, sizeof(float)*size);
    if (err != hipSuccess) CUDAERROR(err);
    s = size;
    j0 = j;
    alpha = a;

    for (int i = 0; i < s; i++) {
        uniform[i] = (float) i/(float) (s-1);
    }
}

// Generate ICDF values
void generateICDF() {
    int r;
    hipError_t err = hipMallocManaged(&invGaus, sizeof(float*)*s);
    if (err != hipSuccess) CUDAERROR(err);
    for (int i = 0; i < s; i++) {
        err = hipMallocManaged(&invGaus[i], sizeof(float)*s);
        if (err != hipSuccess) CUDAERROR(err);
        for (int j = 0; j < s; j++) {
            r = rand()%s;
            invGaus[i][j] = gaussianICDF(uniform[r], i);
        }
    }
}

float getSampleNumber(int i, int j) {
    int r = rand()%s;

    return invGaus[abs(i-j)][r];
}

void freeDistributions() {
    hipFree(uniform);
    for (int i = 0; i < s; i++) {
        hipFree(invGaus[i]);
    }
    hipFree(invGaus);
}
