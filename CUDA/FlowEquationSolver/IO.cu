#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "IO.h"

void readInput(const char *fileName, Var *v) {
    FILE *fp;
    char buff[255];
    char *token;
    if ((fp = fopen(fileName, "r")) == NULL) {
        fprintf(stderr, "Error: No filename found\n");
        exit(1);
    } else {
        while (fgets(buff, 255, fp) != NULL) {
            token = strtok(buff, " ");
            switch (buff[0]) {
                case 'W':
                    token = strtok(NULL, " ");
                    v->W = atof(token);
                    break;
                case 'J':
                    token = strtok(NULL, " ");
                    v->J = atof(token);
                    break;
                case 'h':
                    token = strtok(NULL, " ");
                    v->h = atof(token);
                    break;
                case 'R':
                    token = strtok(NULL, " ");
                    v->R = atoi(token);
                    break;
                case 'S':
                    token = strtok(NULL, " ");
                    v->steps = atoi(token);
                    break;
                case 'N':
                    for (int i = 0; i < v->R; i++) {
                        token = strtok(NULL, " ");
                        v->N[i] = atoi(token);
                    }
                    break;
            }
        }

        v->index = 0;
        fclose(fp);
    }
}

void readArgs(int argc, char *argv[], Var *v) {
    if (argc != 6) {
        fprintf(stderr, "Error: Incorrect number of arguments\n");
        fprintf(stderr, "Usage: main <W> <J> <h> <S> <N>\n");
        exit(1);
    }
    v->W = atof(argv[1]);
    v->J = atof(argv[2]);
    v->h = atof(argv[3]);
    v->R = 1;
    v->steps = atoi(argv[4]);
    v->N[0] = atoi(argv[5]);
    v->index = 0;
}

void outputData (const char *fileName, int *x, double *y, int len) {
    FILE *fp;
    char str[255];
    const char *dir = "data/";
    const char *ext = ".txt";

    strcpy(str, dir);
    strcat(str, fileName);
    strcat(str, ext);

    if ((fp = fopen(str,"w+")) == NULL) {
        fprintf(stderr, "Error: File cannot be created\n");
        exit(-1);
    }

    for (int i = 0; i < len; i++) {
        fprintf(fp,"%d,%f\n", x[i], y[i]);
    }
    fclose(fp);
}

void outputData (const char *fileName, double y) {
    FILE *fp;
    char str[255];
    const char *dir = "data/";
    const char *ext = ".txt";

    strcpy(str, dir);
    strcat(str, fileName);
    strcat(str, ext);

    if ((fp = fopen(str,"w+")) == NULL) {
        fprintf(stderr, "Error: File cannot be created\n");
        exit(-1);
    }

    fprintf(fp,"%f\n", y);
    fclose(fp);
}

void outputHistoryMatrices(const char *fileName, float ***hist, int len, int n) {
    FILE *fp;
    char str[255];
    const char *dir = "data/";
    const char *ext = ".txt";
    strcpy(str, dir);
    strcat(str, fileName);
    strcat(str, ext);

    if ((fp = fopen(str,"w+")) == NULL) {
        fprintf(stderr, "Error: File cannot be created\n");
        exit(-1);
    }

    for (int i = 0; i < len; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                fprintf(fp,"%f",hist[i][min(j,k)][abs(j-k)]);
                if (k != n-1) fprintf(fp,",");
            }
            fprintf(fp,"\n");
        }
    }
    fclose(fp);
}

void outputDiag(const char *fileName, float ***hist, int len, int n) {
    FILE *fp;
    char str[255];
    const char *dir = "data/";
    const char *ext = ".txt";
    strcpy(str, dir);
    strcat(str, fileName);
    strcat(str, ext);

    if ((fp = fopen(str,"w+")) == NULL) {
        fprintf(stderr, "Error: File cannot be created\n");
        exit(-1);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < len; j++) {
            fprintf(fp,"%f",hist[j][i][0]);
            if (j < len -1) fprintf(fp,",");
        }
        fprintf(fp,"\n");
    }
    fclose(fp);
}

void outputElements(const char *fileName, float ***hist, int len, int n) {
    FILE *fp;
    char str[255];
    const char *dir = "data/";
    const char *ext = ".txt";

    strcpy(str, dir);
    strcat(str, fileName);
    strcat(str, ext);

    if ((fp = fopen(str,"w+")) == NULL) {
        fprintf(stderr, "Error: File cannot be created\n");
        exit(-1);
    }

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n-i; j++) {
            fprintf(fp,"{%d,%d},",i,j);
            for (int k = 0; k < len; k++) {
                fprintf(fp,"%f",hist[k][i][j]);
                if (k != n-1) fprintf(fp,",");
            }
            fprintf(fp,"\n");
        }
    }
    fclose(fp);
}

void printVar(Var *var) {
    printf("W: %f\n", var->W);
    printf("J: %f\n", var->J);
    printf("h: %f\n", var->h);
    printf("R: %d\n", var->R);
    printf("N:");
    for (int i = 0; i < var->R; i++) { printf(" %d", var->N[i]); }
    printf("\n");
    printf("steps: %d\n", var->steps);
}
