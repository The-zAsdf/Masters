#include "hip/hip_runtime.h"
#include "measureTime.h"

clock_t start;
clock_t end;

void startTime() {
    start = clock();
}

void endTime() {
    end = clock();
}

double runTime() {
    return (double)(end - start) / CLOCKS_PER_SEC;
}
