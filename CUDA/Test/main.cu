
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE	1024

void VectorAdd(int *a, int *b, int *c, int n)
{
	int i;

	for (i=0; i < n; ++i)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd(a, b, c, SIZE);

	for (int i = 0; i < 10; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
